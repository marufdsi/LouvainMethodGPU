#include "hip/hip_runtime.h"
/*

    Copyright (C) 2016, University of Bergen

    This file is part of Rundemanen - CUDA C++ parallel program for
    community detection

    Rundemanen is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Rundemanen is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Rundemanen.  If not, see <http://www.gnu.org/licenses/>.
    
    */

#include <algorithm>
#include <iostream>
#include "communityGPU.h"

#include <functional>
#include"numeric"

Community::Community(const GraphHOST& input_graph, int nb_pass, double min_mod) {


    //Graph
    g.nb_links = input_graph.nb_links;
    g.nb_nodes = input_graph.nb_nodes;
    g.type = UNWEIGHTED;



    //Copy degree array into indices with an extra zero(0) at the beginning
    g.indices = thrust::device_vector<int>(input_graph.nb_nodes + 1, 0);
    thrust::copy(input_graph.degrees.begin(), input_graph.degrees.end(), g.indices.begin() + 1); // 0 at first position

    /********************Gather Graph Statistics***************/
    std::vector< int> vtxDegs;

    vtxDegs.resize(input_graph.degrees.size());

    std::adjacent_difference(input_graph.degrees.begin(), input_graph.degrees.end(), vtxDegs.begin());

    int totNbrs = std::accumulate(vtxDegs.begin(), vtxDegs.end(), 0);
    int maxDeg = *std::max_element(vtxDegs.begin(), vtxDegs.end());

    double sumSquareDiff = 0;
    double avgDeg = (double) totNbrs / g.nb_nodes;

    for (int i = 0; i < vtxDegs.size(); i++) {
        double delta = ((double) vtxDegs[i] - avgDeg);
        sumSquareDiff += delta*delta;
    }

    double standardDeviation = sqrt(sumSquareDiff / input_graph.nb_nodes);

    std::cout << "MaxDeg = " << maxDeg << " AvgDeg = " << avgDeg << " STD = "
            << standardDeviation << " STD2AvgRatio = " << standardDeviation / avgDeg << std::endl;

    std::cout << "totNbrs =" << totNbrs << " #links =" << input_graph.nb_links << std::endl;

    if (input_graph.nb_nodes < 10) {
        std::cout << std::endl;
        for (int i = 0; i < vtxDegs.size(); i++) {
            std::cout << vtxDegs[i] << " ";
        }
        std::cout << std::endl;
    }
    /**********************************/

    //copy all edges
    g.links.resize(g.nb_links);
    g.links = input_graph.links;

    //copy all weights
    g.weights.resize(input_graph.weights.size());
    g.weights = input_graph.weights;

    std::cout << std::endl << "Copied  " << g.weights.size() << " weights" << std::endl;

    g.total_weight = input_graph.total_weight;


    if (input_graph.weights.size() > 0) {
        g.type = WEIGHTED;
        std::cout << " Setting type to WEIGHTED" << std::endl;
    } else {
        std::cout << "Type is already set to UNWEIGHTED" << std::endl;
    }

    //Community
    community_size = g.nb_nodes;
    min_modularity = min_mod;

    std::cout << std::endl << "(Dev Graph) " << " #Nodes: " << g.nb_nodes << "  #Links: " << g.nb_links / 2 << "  Total_Weight: " << g.total_weight / 2 << std::endl;
    std::cout << "community_size: " << community_size << std::endl;
    // seriously !!
}
