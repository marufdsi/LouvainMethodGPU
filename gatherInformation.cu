#include "hip/hip_runtime.h"
/*

    Copyright (C) 2016, University of Bergen

    This file is part of Rundemanen - CUDA C++ parallel program for
    community detection

    Rundemanen is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Rundemanen is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Rundemanen.  If not, see <http://www.gnu.org/licenses/>.
    
    */


#include"fstream"
#include"communityGPU.h"

template < class T>
__global__
void filter_entries_by_threshold(T *source, T* dest, T threshold, int nr_old_communities, int* locations) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < nr_old_communities) {
        // If source[tid] contains something > threshold  and 
        // locations[i] is greater than zero(0)
        T data = source[tid];
        int index = locations[tid];

        if (data > threshold && index >= 0) { //zero based indexing
            //printf("\n %d %d %d\n", tid, index, data);
            dest[index] = source[tid];
        }

        tid += blockDim.x * gridDim.x;
    }
}

void Community::gatherStatistics(bool isPreprocess) {

    bool hostPrint = false;
    int sc;
    sc = 0; //std::cin>>sc;
    hostPrint = (sc > 1);

    thrust::device_vector<int> renumber(community_size, 0);

/*
    thrust::host_vector<int> hn2c = n2c;
    std::ofstream ofs;
    ofs.open ("n2c.txt", std::ofstream::out | std::ofstream::app);
    for(int i=0; i< hn2c.size(); i++) {

	ofs<<i<<":"<<hn2c[i]<<" ";

	if(hn2c[i]==320)
		std::cout<<"####" << i  <<"; (cid before renumbering) "<<hn2c[i]<<std::endl;

	if( (i+1)%100 == 0)
		ofs<<"\n";		
    }

    ofs<<"\n";
    ofs.close();
*/
    if (hostPrint) {
        print_vector(renumber, " Size of Communities: ");
        print_vector(n2c, "n2c: ");
    }

    int load_per_blk = CHUNK_PER_WARP * (NR_THREAD_PER_BLOCK / PHY_WRP_SZ);
    int nr_of_block = (community_size + load_per_blk - 1) / load_per_blk;


    //Count the size of each new community and store the sizes in renumber

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    if(isPreprocess ==false ){
    get_size_of_communities << < nr_of_block, NR_THREAD_PER_BLOCK >>>(
            thrust::raw_pointer_cast(renumber.data()), thrust::raw_pointer_cast(n2c.data()), g.nb_nodes);
    }else{
    get_size_of_communities_NEW << < nr_of_block, NR_THREAD_PER_BLOCK >>>(
            thrust::raw_pointer_cast(renumber.data()), thrust::raw_pointer_cast(n2c.data()), g.nb_nodes,thrust::raw_pointer_cast(g.indices.data()));
    }
    report_time(start, stop, "get_size_of_communities");


    if (hostPrint) {
        print_vector(renumber, "Size of Communities: (by Atomic Add) ");
    }


    n2c_new.resize(community_size, 0);// DO NOT clear it here, it will be cleared in gatherStatistics
    // 1 to indicate community of size greater or equal 1, 

    assert(renumber.size() == n2c_new.size());
  
    //int myZero = 0;
    //if(isPreprocess == true)
	//myZero = 1;

    thrust::transform(thrust::device, renumber.begin(), renumber.end(),
            n2c_new.begin(), IsGreaterThanZero<int>(0));

    //NOTE:n2c_new contains 0s and 1s only

    if (hostPrint) {
        print_vector(n2c_new, "Before Prefix Sum: ");
    }

    thrust::inclusive_scan(thrust::device, n2c_new.begin(), n2c_new.end(), n2c_new.begin());

    int new_nb_comm = n2c_new.back();

    if (hostPrint) {
        print_vector(n2c_new, "After Prefix Sum: ");
    }


    thrust::transform(thrust::device, renumber.begin(), renumber.end(),
            n2c_new.begin(), n2c_new.begin(), Community_ID_By_Prefix_Sum<int>());

    // After Transform, n2c_new contains mapping from old_CId to new_Cid
    {

        //(renumber): 0 4 0 0 0, 0 4 0 0 0,......( #member per Community)
        // (n2c_new): 0 1 1 1 1, 1 2 2 2 2,...............   

        //( n2c_new): X 0 X X X, X 1 X X X,.......( New CId, X = -1 in my case)

        //n2c_new now holds renumbered CIDs *******
    }

    /*
    thrust::host_vector<int> hn2c_new = n2c_new;
   
    	for(int i=0; i< hn2c_new.size(); i++) {
    		if(hn2c_new[i] == 63 || ( i>=320 && i<=329) || (i >=2945 && i <=2949) )
			std::cout<<"node:"<<i<<" ; new_CID "<<hn2c_new[i]<<std::endl;
	}
    */
    
    if (hostPrint) {
        print_vector(n2c_new, "CID by Prefix Sum: ");
    }

   //std::cout << "#NewCommunity: " << new_nb_comm << std::endl;

    //for next phase
    pos_ptr_of_new_comm.resize(new_nb_comm + 1, 0);



    if (hostPrint) {
        print_vector(pos_ptr_of_new_comm, "Pos ptrs: ");
    }

    // copy all non zero entries that corresponds to #nodes per communities 

    //Note: renumber still contains #nodes per community
    //n2c_new contains renumbered cIds of communities having size >=1

    hipEventRecord(start, 0);

    filter_entries_by_threshold << < nr_of_block, NR_THREAD_PER_BLOCK >>>(
            thrust::raw_pointer_cast(renumber.data()), thrust::raw_pointer_cast(pos_ptr_of_new_comm.data()) + 1,
            (int) 0, community_size, thrust::raw_pointer_cast(n2c_new.data()));

    report_time(start, stop, "filter_entries_by_threshold");

    // copied all non zero entries that corresponds to #nodes per communities into pos_ptr_of_new_comm

    if (hostPrint) {
        print_vector(pos_ptr_of_new_comm, "Pos ptrs: ");
    }

    // after prefix sum pos_ptr points to start of each community where 
    //nodes of same community are placed consecutively

    thrust::inclusive_scan(thrust::device, pos_ptr_of_new_comm.begin(),
            pos_ptr_of_new_comm.end(), pos_ptr_of_new_comm.begin(),
            thrust::plus<int>());


    if (hostPrint) {
        print_vector(pos_ptr_of_new_comm, "Pos ptrs: ");

    }

    g_next.nb_nodes = new_nb_comm;

    renumber.clear();

}
